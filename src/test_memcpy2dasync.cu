#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "./common.h"

/*
 * Check whether hipMemcpy2DAsync uses row major or colmun major.
 */

#define NSTREAM 1

void initialData(float *ip, int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void checkResult(float *hostRef, float *gpuRef, int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}

int main(int argc, char *argv[])
{
    printf("> %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // set up max connectioin
    char * iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "8", 1);
    char *ivalue =  getenv (iname);
    printf ("> %s = %s\n", iname, ivalue);
    printf ("> with streams = %d\n", NSTREAM);

    // set up testing
    int m = 3;
    int n = 2;
    int dpitch = 2; // row major
    int spitch = 3; // row major

    // The test matrix A is:
    // |  7  | 8  | 9  | 
    // |  10 | 11 | 12 | 
    // |  13 | 14 | 15 | 
    // |  16 | 17 | 18 | 
    // test store in row major
    float h_A[] = {7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18};

    // The sub-matrix we expect:
    // |  7 | 8  |
    // | 10 | 11 |
    // | 13 | 14 |
    // test store in row major
    float expected_gpuRef[] = {7, 8, 10, 13, 14};

    // malloc host memory
    float *gpuRef = (float *)malloc(m * n * sizeof(float));
    // initialize data at host side
    memset(gpuRef, 0, m * n * sizeof(float));

    // malloc device global memory
    float *d_MatA;
    hipMalloc((void **)&d_MatA, m * n * sizeof(float));

    // transfer data from host to device
    CHECK(hipMemcpy2DAsync(d_MatA, dpitch, h_A, spitch, n, m, hipMemcpyHostToDevice, 0));

    // copy memcpy result back to host side
    hipMemcpy(gpuRef, d_MatA, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // check device results
    checkResult(expected_gpuRef, gpuRef, m * n);

    // free device global memory 
    hipFree(d_MatA); 

    // free host memory 
    free(gpuRef);
}