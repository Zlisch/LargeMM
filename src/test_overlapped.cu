#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "./common.h"

#define NSTREAM 4

void checkResult(float *hostRef, float *gpuRef, int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");
}

void printMatrix(float *C, int N)
{
    printf("\n");
    for (int i = 0; i < N; i++)
    {
        printf("(%f)\b\b", C[i]);
    }
    printf("\n");
}

int main(int argc, char *argv[]) 
{
    printf("> %s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // set up max connectioin
    char * iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    setenv (iname, "8", 1);
    char *ivalue =  getenv (iname);
    printf ("> %s = %s\n", iname, ivalue);
    printf ("> with streams = %d\n", NSTREAM);

    // set up data size of matrix 
    int m = 4;
    int k = 4;
    int n = 4;
    printf("MM Size: m = %d, k = %d, n = %d\n", m, k, n);

    // calculalte data size of one square matix
    int nElem = m * n;
    printf("> square matrix size = %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // The test matrix A is:
    // |  1  | 2  | 3  | 4  |
    // |  5  | 6  | 7  | 8  |
    // |  9  | 10 | 11 | 12 | 
    // |  13 | 14 | 15 | 16 | 
    // test store in row major
    float h_A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};

    // The test matrix ABis:
    // |  4  | 2  | 3  | 4  |
    // |  5  | 1  | 7  | 1  |
    // |  9  | 10 | 17 | 12 | 
    // |  13 | 14 | 15 | 10 | 
    // test store in row major
    float h_B[] = {4, 2, 3, 4, 5, 1, 7, 1, 9, 10, 17, 12, 13, 14, 15, 10};

    // The result we expect:
    // |  93  | 90  | 128 | 82  |
    // |  217 | 198 | 296 | 190 |
    // |  341 | 306 | 464 | 298 | 
    // |  465 | 414 | 632 | 406 | 
    // should be stored in col major
    float expected_gpuRef[] = {93, 217, 341, 465, 90, 198, 306, 414, 128, 296, 464, 632, 82, 190, 298, 406};

    // malloc host memory
    float *gpuRef = (float *)malloc(m * n * sizeof(float));
    // initialize data at host side
    memset(gpuRef, 0, m * n * sizeof(float));

    // initialize CUBLAS context
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context
    stat = hipblasCreate(&handle); 

    float alpha = 1.0f;
    float beta = 0.0f;

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((void **)&d_MatA, m * k * sizeof(float)); 
    hipMalloc((void **)&d_MatB, k * n * sizeof(float)); 
    hipMalloc((void **)&d_MatC, m * n * sizeof(float)); 

    // for synchronize the streams and the default stream
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // initialise streams
    hipStream_t stream[NSTREAM];
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }

    CHECK(hipEventRecord(start, 0));

    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i)
    {
        if (i == 0 || i == 2) 
            hipMemcpy2DAsync(d_MatA + (i/2) * (nElem/2), k * sizeof(float), h_A + (i/2) * (nElem/2), k * sizeof(float), k * sizeof(float), m/2, hipMemcpyHostToDevice, stream[i]);
        if (i == 0 || i == 1)
            hipMemcpy2DAsync(d_MatB + (i%2) * (n/2), n * sizeof(float), h_B + (i%2) * (n/2), n * sizeof(float), (n/2) * sizeof(float), k, hipMemcpyHostToDevice, stream[i]);
        stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m/2, n/2, k, &alpha, d_MatA + (i/2) * (nElem/2), k, d_MatB + (i%2) * (n/2), n, &beta, d_MatC + (i%2) * (nElem/2) + (i/2) * (n/2), n);
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // transfer data from host to device
    // general case for A
    // CHECK(hipMemcpy2DAsync(d_MatA + (3/2) * (nElem/2), k * sizeof(float), h_A + (3/2) * (nElem/2), k * sizeof(float), k * sizeof(float), m/2, hipMemcpyHostToDevice, 0));
    // general case for B
    // CHECK(hipMemcpy2DAsync(d_MatB + (3%2) * (n/2), n * sizeof(float), h_B + (3%2) * (n/2), n * sizeof(float), (n/2) * sizeof(float), k, hipMemcpyHostToDevice, 0););

    // copy memcpy result back to host side
    hipMemcpy(gpuRef, d_MatC, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // check device results
    // checkResult(expected_gpuRef, gpuRef, m * n);

    // print returned matrix
    printf("The gpu matrix:\n");
    printMatrix(gpuRef, 4*4);

    // free device global memory 
    hipFree(d_MatA); 
    hipFree(d_MatB); 
    hipFree(d_MatC);

    // destroy CUBLAS context
    hipblasDestroy(handle);

    // free host memory 
    free(gpuRef);

    return EXIT_SUCCESS;
}